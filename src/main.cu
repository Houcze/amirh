#include "hip/hip_runtime.h"
#include <tensor.h>
#include <iostream>
#include <cstdlib>
#include <vector>
#define d 1
#define dx 1
#define dy 1
double omega = 0.0007292;


void Jpp(io::cuda::tensor *A, io::cuda::tensor *B, io::cuda::tensor *output)
{
	*output = (1 / (4 * d * d)) * (
		(bias(*A, 0, 1) - bias(*A, 0, -1)) * (bias(*B, -1, 0) - bias(*B, 1, 0)) - 
		(bias(*A, -1, 0)  - bias(*A, 1, 0)) * (bias(*B, 0, 1) - bias(*B, 0, -1)) 
	);
	std::cout << "Here!!!" << std::endl;
}

void Jpm1(io::cuda::tensor *A, io::cuda::tensor *B, io::cuda::tensor *output)
{
	*output = (1 / (4 * d * d)) * (
		bias(*A, -1, 1) * (bias(*B, -1, 0) - bias(*B, 0, 1)) -
		bias(*A, 1, -1) * (bias(*B, 0, -1) - bias(*B, 1, 0)) + 
		bias(*A, -1, -1) * (bias(*B, 0, -1) - bias(*B, -1, 0)) - 
		bias(*A, 1, 1) * (bias(*B, 1, 0) - bias(*B, 0, 1))
	);
}

void Jmp1(io::cuda::tensor *A, io::cuda::tensor *B, io::cuda::tensor *output)
{
	*output = (1 / (4 * d * d)) * (
		bias(*A, 0, 1) * (bias(*B, -1, 1) - bias(*B, 1, 1)) - 
		bias(*A, 0, -1) * (bias(*B, -1, -1) - bias(*B, 1, -1)) - 
		bias(*A, -1, 0) * (bias(*B, -1, 1) - bias(*B, -1, -1)) + 
		bias(*A, 1, 0) * (bias(*B, 1, 1) - bias(*B, 1, -1))
	);
}

void Jmm(io::cuda::tensor *A, io::cuda::tensor *B, io::cuda::tensor *output)
{
	*output = (1 / (8 * d * d)) * (
		(bias(*A, -1, 1) - bias(*A, 1, -1)) * (bias(*B, -1, -1) - bias(*B, 1, 1)) -
		(bias(*A, -1, -1) - bias(*A, 1, 1)) * (bias(*B, -1, 1) - bias(*B, 1, -1))
	);
}

void Jmp2(io::cuda::tensor *A, io::cuda::tensor *B, io::cuda::tensor *output)
{
	*output = (1/ (8 * d * d)) * (
		bias(*A, -1, 1) * (bias(*B, -2, 0) - bias(*B, 0, 2)) - 
		bias(*A, 1, -1) * (bias(*B, 0, -2) - bias(*B, 0, 1)) - 
		bias(*A, -1, -1) * (bias(*B, -2, 0) - bias(*B, 0, -2)) + 
		bias(*A, 1, 1) * (bias(*B, 0, 2) - bias(*B, 2, 0)) 
	);
}

void Jpm2(io::cuda::tensor *A, io::cuda::tensor *B, io::cuda::tensor *output)
{
	*output = (1 / ( 8 * d * d)) * (
        bias(*A, -2, 0) * (bias(*B, -1, -1) - bias(*B, -1, 1)) -
        bias(*A, 2, 0) * (bias(*B, 1, -1) - bias(*B, 1, 1)) -
        bias(*A, 0, 2) * (bias(*B, 1, -1) - bias(*B, -1, -1)) +
        bias(*A, 0, -2) * (bias(*B, 1, 1) - bias(*B, -1, 1))	
	);
}

void J1(io::cuda::tensor *A, io::cuda::tensor *B, io::cuda::tensor *output)
{
	io::cuda::tensor ja = *output;
	io::cuda::tensor jb = *output;
	io::cuda::tensor jc = *output;
	Jpp(A, B, &ja);
	Jpm1(A, B, &jb);
	Jmp1(A, B, &jc);
	*output = (1 / 3) * (ja + jb + jc);
}

void J2(io::cuda::tensor *A, io::cuda::tensor *B, io::cuda::tensor *output)
{
	io::cuda::tensor ja = *output;
	io::cuda::tensor jb = *output;
	io::cuda::tensor jc = *output;
	Jmm(A, B, &ja);
	Jmp2(A, B, &jb);
	Jpm2(A, B, &jc);
	*output = (1 / 3) * (ja + jb + jc);
}


void J(io::cuda::tensor *A, io::cuda::tensor *B, io::cuda::tensor *output)
{
	io::cuda::tensor j1 = *output;
	io::cuda::tensor j2 = *output;
	J1(A, B, &j1);
	J2(A, B, &j2);
	*output = 2 * j1 - j2;
}

void zeta(io::cuda::tensor *u, io::cuda::tensor *v, io::cuda::tensor *output)
{
	// \zeta = (\partial{v}) / (\partial{x}) - (\partial{u}) / (\partial{y})
	// in wind format
	io::cuda::tensor v_x = (3 * (*v) - 4 * bias(*v, -1, 0) + bias(*v, -2, 0)) / (2 * dy);
	io::cuda::tensor u_y = (3 * (*u) - 4 * bias(*u, 0, -1) + bias(*u, 0, -2)) / (2 * dx);
	*output = v_x - u_y;
}



int main(int argc, char* argv[])
{
	io::cuda::tensor a{argv[1], argv[2]};
	size_t size;
	size_t dims;
	size = a.get_size();
	dims = a.get_dims();
	size_t * shape;
	shape = (size_t *) std::malloc(dims * sizeof(size_t));
	shape = a.get_shape();
	io::cuda::tensor output{size, dims, shape};
	J(&a, &a, &output);
	auto b = cos(a);
	double *data;
	data = (double *) std::malloc(size * sizeof(double));
	int status = b.dataSync(data);	
	for(int i=0; i<a.get_size(); i++)
	{
		std::cout << *(data + i) << '\t';
	}
	std::cout << "I am J here!!" << std::endl;
			
}
