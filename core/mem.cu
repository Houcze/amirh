#include <core/mem.h>
#include <iostream>
#include <hip/hip_runtime.h>

double *Nallocator::register_variable(std::string vname)
{
    double *v;
    hipMalloc(&v, Prop::size(NOSH) * sizeof(double));
    vmap[vname] = v;
    vargs[vname] = NOSH;
    return v;
}

double *Nallocator::register_variable(std::string vname, Prop::shape s)
{
    double *v;
    hipMalloc(&v, sizeof(double) * Prop::size(s));
    vmap[vname] = v;
    vargs[vname] = s;
    return v;
}

void Nallocator::print_vmap_length()
{
    std::cout << vmap.size() << std::endl;
}

double *Nallocator::require_variable(std::string vname)
{
    return vmap[vname];
}

bool Nallocator::isn(std::string vname)
{
    return vmap.count(vname);
}

int Nallocator::deallocate_variable(double *v, std::string vname)
{
    hipFree(v);
    vmap.erase(vmap.find(vname));
    return EXIT_SUCCESS;
}

int Nallocator::register_shape(Prop::shape s_)
{
    NOSH = s_;
    return EXIT_SUCCESS;
}

int Nallocator::register_seqlen(int len)
{
    for (int i = 0; i < len; i++)
    {
        register_variable("v" + std::to_string(i));
    }
    return EXIT_SUCCESS;
}

void Nallocator::print_variable_list()
{
    for (auto it : vmap)
    {
        std::cout << it.first << std::endl;
    }
}

Prop::shape Nallocator::registered_shape()
{
    return NOSH;
}
